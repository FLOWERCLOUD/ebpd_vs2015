#include "hip/hip_runtime.h"
#include "cuda_rendering.hpp"

#include "toolbox/gl_utils/gltex2D.hpp"
#include "animesh.hpp"
#include "toolbox/gl_utils/glbuffer_object.hpp"
#include "depth_peeling.hpp"
#include "opengl_stuff.hpp"
#include "cuda_globals.hpp"
#include "cuda_stuff.hpp"
#include "cuda_ctrl.hpp"
#include "skeleton.hpp"
#include "filters.hpp"
#include "globals.hpp"
#include "g_vbo_primitives.hpp"
#include "scene.hpp"

#include "cuda_main_kernels.hpp"
#include "toolbox/gl_utils/gldirect_draw.hpp"

#ifndef M_PI
#define M_PI (3.14159265358979323846f)
#endif

// Forward def "skeleton_env.hpp" ----------------------------------------------
namespace Skeleton_env {
void draw_grids();
}
// END Forward def "skeleton_env.hpp" ------------------------------------------

// -----------------------------------------------------------------------------

void draw_controller()
{
    const std::vector<int>& set = Cuda_ctrl::_skeleton.get_selection_set();
    if(set.size() > 0)
    {
        int bone_id = set[set.size()-1];
        int pt = g_skel->parent( bone_id );
        if( pt > -1 ){
            Blending_env::Ctrl_id ctrl_id = g_skel->get_ctrl(/*bone_id*/ pt);
            draw_controller(ctrl_id, 0, 0, 320, 240);
        }
    }
    else
    {
        glColor3f(0.f, 1.f, 0.f);
        draw_global_controller(0  , 0, 320, 240);
    }
}

// -----------------------------------------------------------------------------

static void draw_junction_sphere(bool rest_pose)
{
    if(g_skel == 0 || !Cuda_ctrl::_display._junction_spheres)
        return;

    using namespace Cuda_ctrl;
    glColor4f(0.4f, 0.3f, 1.f, 0.5f);
    const std::vector<int>& set = _skeleton.get_selection_set();
    for(unsigned i = 0; i < set.size(); i++)
    {
        float r  = g_animesh->get_junction_radius(set[i]);
        Vec3 v = rest_pose ? g_skel->joint_rest_pos(set[i]) : g_skel->joint_pos(set[i]);
        glPushMatrix();
        glTranslatef(v.x, v.y, v.z);
        glScalef(r, r, r);
        g_primitive_printer.draw(g_sphere_vbo);
        glPopMatrix();
    }
}

// -----------------------------------------------------------------------------

static void draw_bbox()
{
    if(!Cuda_ctrl::_display._oriented_bbox && !Cuda_ctrl::_display._aa_bbox)
        return;

    using namespace Cuda_ctrl;
    const std::vector<int>& selection_set = _skeleton.get_selection_set();
    _color.get(Color_ctrl::BOUNDING_BOX).set_gl_state();
    for(unsigned int i = 0; i < selection_set.size(); i++)
    {
        int id = selection_set[i];
        const Bone* b = g_skel->get_bone(id);

        if(b->get_type() == EBone::SSD)
            continue;

        Obbox obbox = b->get_obbox();

        Vec3 lengths = obbox._bb.lengths();
        if(Cuda_ctrl::_display._oriented_bbox)
        {
            glPushMatrix();

            glMultMatrixf(obbox._tr.transpose().m);
            glTranslatef(obbox._bb.pmin.x, obbox._bb.pmin.y, obbox._bb.pmin.z);
            glScalef(lengths.x, lengths.y, lengths.z);

            glColor3f(0.f, 0.f, 0.f);
            g_primitive_printer.draw(g_cube_vbo);
            glPopMatrix();
        }

        if( Cuda_ctrl::_display._aa_bbox )
        {
            Bbox3 bbox = b->get_bbox();

            lengths = bbox.lengths();
            glPushMatrix();


            glTranslatef(bbox.pmin.x, bbox.pmin.y, bbox.pmin.z);
            glScalef(lengths.x, lengths.y, lengths.z);

            glColor3f(1.f, 1.f, 0.f);
            g_primitive_printer.draw( g_cube_vbo );
            glPopMatrix();
        }
    }
}

// -----------------------------------------------------------------------------

static void draw_cylinder()
{
    using namespace Cuda_ctrl;
    const std::vector<int>& selection_set = _skeleton.get_selection_set();

    for(unsigned int i = 0; i<selection_set.size(); i++)
    {
        int id = selection_set[i];
        if( g_skel->bone_type(id) == EBone::CYLINDER )
        {
            const Bone* b = g_skel->get_bone(id);
            float rad = b->radius();
            glMatrixMode(GL_MODELVIEW);

            glPushMatrix();
            Transfo tr_trans = b->get_frame().transpose();
            glMultMatrixf(tr_trans.m);
            glRotatef(90.f, 0.f, 1.f, 0.f);
            glScalef(rad, rad, b->length());
            g_primitive_printer.draw(g_cylinder_cage_vbo);
            glPopMatrix();
        }

    }
}

// -----------------------------------------------------------------------------

static void draw_hrbf_points(bool rest_pose)
{
    using namespace Cuda_ctrl;

    if(_display._draw_hrbf_samples ||
        _display._edit_hrbf_samples)
    {
        const std::vector<int>& selection_set = _skeleton.get_selection_set();
        if(_anim_mesh != 0)
            _anim_mesh->draw_hrbf_points(selection_set, true, rest_pose);
    }
}

// -----------------------------------------------------------------------------

static void draw_mesh_points(const Camera* cam, bool rest_pose)
{
    if(Cuda_ctrl::_anim_mesh == 0) return;

    if(Cuda_ctrl::_anim_mesh->is_point_displayed())
    {
        // Do a little offset so mesh points won't hide rbf samples
        const float eps = 1.0001f;
        glPushMatrix();
        Vec3 p = cam->get_pos();
        glTranslatef(p.x, p.y, p.z);
        glScalef(eps, eps, eps);
        glTranslatef(-p.x, -p.y, -p.z);

        glPointSize(9.f);
        GLEnabledSave save_point (GL_POINT_SMOOTH, true, true );
        rest_pose ? g_animesh->draw_points_rest_pose() : g_mesh->draw_points();
        glPopMatrix();
    }
}

// -----------------------------------------------------------------------------

void draw_normals(const std::vector<int>& selected_points,
                  const Cuda_utils::DA_Vec3& d_ssd_normals)
{
    Vec3* vert = 0;

    g_mesh->_mesh_gl._vbo.map_to(vert, GL_READ_ONLY);

    glBegin(GL_LINES);
    glColor4f(1.f, 0.f, 0.f, 1.f);
    for(unsigned i = 0; i < selected_points.size(); i++)
    {
        Vec3 n;
        d_ssd_normals.fetch(selected_points[i], n);
        n.normalize();
        n = n * 5.f;

        const EMesh::Packed_data d = g_mesh->get_packed_vert_map()[selected_points[i]];
        Vec3 v = vert[ d._idx_data_unpacked ];

        //glColor3f(n.x, n.y, n.z);
        glVertex3f(v.x, v.y, v.z);
        glVertex3f(v.x + n.x, v.y + n.y, v.z + n.z);

    }
    glEnd();

    if( g_animesh ) g_animesh->hd_verts_rots.update_host_mem();
    for(unsigned i = 0; i < selected_points.size(); i++)
    {
        if( g_animesh )
        {
            Mat2 r = g_animesh->hd_verts_rots[selected_points[i]];
            ((r+r)*0.5f).print();

            std::cout << "bla: "<< r.m[1] << std::endl;

            r = Mat2::rotate( -asinf(r.m[1]) );
            ((r+r)*0.5f).print();

            std::cout << std::endl;
        }
    }

    g_mesh->_mesh_gl._vbo.unmap();
}

// -----------------------------------------------------------------------------

// TODO: re-write this with using context
void redraw_with_ssao(/* Render_context_cu* ctx */)
{
#if 0
    const int width  = Cuda_ctrl::_display._width;
    const int height = Cuda_ctrl::_display._height;

    ssao_shader->use();
    GLEnabledSave texture_2d(GL_TEXTURE_2D, true, true);
    GLActiveTexUnitSave tex_unit_save;

    glActiveTexture(GL_TEXTURE0);
    glAssert( glBindTexture(GL_TEXTURE_2D, gl_Tex[NORMAL_MAP]) );
    glCopyTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA,
                     0, 0, width, height, 0);

    ssao_shader->set_uniform("bgl_RenderedTexture", 0);


    glActiveTexture(GL_TEXTURE2);
    glAssert( glBindTexture(GL_TEXTURE_2D, gl_Tex[DEPTH]) );
    glCopyTexImage2D(GL_TEXTURE_2D, 0, GL_DEPTH_COMPONENT24,
                     0, 0, width, height, 0);

    ssao_shader->set_uniform("bgl_DepthTexture", 2);

    int v = Cuda_ctrl::_display._width;
    ssao_shader->set_uniform("width", v);
    v = Cuda_ctrl::_display._height;
    ssao_shader->set_uniform("height", v);

    draw_quad();
    ShaderProgram::unuse();
#endif
}

// -----------------------------------------------------------------------------

void draw_animesh(bool use_color_array, bool use_point_color, bool rest_pose)
{
    if(g_animesh == 0) return;

    if(rest_pose) g_animesh->draw_rest_pose(use_color_array, use_point_color);
    else          g_animesh->draw(use_color_array, use_point_color);
}

// -----------------------------------------------------------------------------

static void draw_skeleton(const Camera* cam, bool draw_skel, bool rest_pose)
{
    using namespace Cuda_ctrl;
    std::vector<int> joints = _skeleton.get_selection_set();
    int              node   = _graph.get_selected_node();

    if     ( draw_skel && g_skel != 0 ) g_skel ->draw( *cam, joints, rest_pose);
    else if( g_graph != 0 )             g_graph->draw( *cam, node );
}

// -----------------------------------------------------------------------------

///////////TODO to be deleted
bool g_test = false;

void draw_wires_selection()
{

    if( !g_test ) return;

    GLLineWidthSave width(3.f);

    const std::vector<int>& set = Cuda_ctrl::_anim_mesh->get_selected_points();

    Vec3* vert = 0;
    g_mesh->_mesh_gl._vbo.map_to( vert, GL_READ_ONLY );

    glBegin(GL_LINE_STRIP);
    glColor3f(0.f, 0.f, 0.f);
    for(unsigned i = 0; i < set.size(); ++i) {
        int idx = set[i];
        EMesh::Packed_data pack = g_mesh->get_packed_vert_map()[idx];
        if( pack._nb_ocurrence > 0 ){
            idx = pack._idx_data_unpacked;
            Vec3 pos = vert[idx];
            glVertex3f(pos.x, pos.y, pos.z);
        }
    }
    glEnd();

    glBegin(GL_POINTS);
    glColor3f(0.f, 0.f, 0.f);
    for(unsigned i = 0; i < set.size(); ++i) {
        int idx = set[i];
        EMesh::Packed_data pack = g_mesh->get_packed_vert_map()[idx];
        if( pack._nb_ocurrence > 0 ){
            idx = pack._idx_data_unpacked;
            Vec3 pos = vert[idx];
            glVertex3f(pos.x, pos.y, pos.z);
        }
    }
    glEnd();

    g_mesh->_mesh_gl._vbo.unmap();

}

// -----------------------------------------------------------------------------

/// what is drawn here will be hidden by the transceluscent mesh
/// Hardware Antialiasing will work
static void plain_objects(const Camera* cam, const Render_context_cu* ctx,
                          float r, float g, float b, float bfactor)
{
    using namespace Cuda_ctrl;
    Cuda_ctrl::_potential_plane.draw();

    if( _display._wire )
    {
        glAssert( glColor4f(r, g, b, bfactor) );
        if( !ctx->_skeleton )
        {
            if ( _anim_mesh != 0 ) draw_animesh(true, true, true /*rest pose*/);
            else if( g_mesh != 0 ) g_mesh->draw(true, true);
        }
        else
        {
            if(ctx->_draw_mesh) draw_animesh(true, true, ctx->_rest_pose);
        }
    }

    if( !ctx->_plain_phong )
    {
        if(g_mesh!= 0 && g_mesh->get_nb_vertices()>0) g_mesh->debug_draw_edges();
        draw_wires_selection();///////////TODO to be deleted
        draw_hrbf_points(ctx->_rest_pose);
        draw_mesh_points(cam, ctx->_rest_pose);
        if( Cuda_ctrl::_display._grid) draw_grid_lines(cam);

        draw_bbox();
        draw_cylinder();

        if(_debug._show_normals)
            draw_normals(_anim_mesh->get_selected_points(),
                         g_animesh->get_ssd_normals());

        if(_debug._show_gradient)
            _debug.draw_gradient( _anim_mesh->get_selected_points(),
                                 g_animesh->get_gradient().d_ptr(),
                                 g_animesh->d_vert_buffer_2.ptr() /*g_animesh->d_grad_transfo.ptr()*/);

    }

}

// -------------------------------------------------------------------------

/// What follows won't benefit from hardware AA, but will be visible in
/// transluscent mode
static void transparent_objects(const Camera* cam, const Render_context_cu* ctx)
{
    using namespace Cuda_ctrl;
    Cuda_ctrl::_potential_plane.draw();


    if( !ctx->_skeleton )
    {
        if     ( _anim_mesh != 0 ) draw_animesh(true, false, true /*rest pose*/);
        else if( g_mesh     != 0 ) g_mesh->draw();
    }
    else
    {
        if(ctx->_draw_mesh) draw_animesh(true, false, ctx->_rest_pose);
    }


    if( !ctx->_plain_phong  && Cuda_ctrl::_skeleton.is_displayed())
    {
        if(_anim_mesh != 0)
            Cuda_ctrl::_anim_mesh->draw_rotation_axis();

        glAssert( glColor4f(1.f, 0.f, 0.f, 1.f) );
        draw_junction_sphere(ctx->_rest_pose);
        draw_skeleton(cam, ctx->_skeleton, ctx->_rest_pose);
    }
}


// Class RenderFuncWireframe ===================================================

void RenderFuncWireframe::draw_transc_objs()
{
    transparent_objects(_cam, _ctx);
}

// -----------------------------------------------------------------------------

void RenderFuncWireframe::f()
{
    //glPolygonOffset(1.1f, 4.f);
    //glEnable(GL_POLYGON_OFFSET_FILL);
    transparent_objects(_cam, _ctx);
    //glDisable(GL_POLYGON_OFFSET_FILL);
}

// -----------------------------------------------------------------------------

void RenderFuncWireframe::render(const Camera* cam,
                                 const Render_context_cu* ctx)
{
    GLEnabledSave save_light(GL_LIGHTING, true, false);
    glAssert( glPolygonMode(GL_FRONT_AND_BACK, GL_LINE) );
    glAssert( glLineWidth(1.f) );
    glAssert( glHint(GL_LINE_SMOOTH_HINT, GL_NICEST) );
    glAssert( glEnable(GL_LINE_SMOOTH) );

    glAssert( glBlendFunc(GL_DST_ALPHA, GL_ONE_MINUS_SRC_ALPHA) );
    //plain_objects(cam, 0.1f, 0.15f, 0.7f, 1.f);
    plain_objects(cam, ctx, 1.f, 1.f, 1.f, 1.f);
    glAssert( glDisable(GL_LINE_SMOOTH) );
    glAssert( glPolygonMode(GL_FRONT_AND_BACK, GL_FILL) );
}

// END RENDER_FUNC_WIRE_FRAME ==================================================

// CLASS Render_context_cu =====================================================

Render_context_cu::Render_context_cu(int w, int h):
    _pbo_color(0),
    _pbo_depth(0),
    _d_img_buffer(0),
    _d_bloom_buffer(0),
    _d_rendu_buf(0),
    _d_rendu_depth_buf(0),
    _plain_phong(false),
    _textures(true),
    _draw_mesh(true),
    _raytrace(false),
    _skeleton(false),
    _rest_pose(false)
{
    _peeler    = new Peeler();
    _frame_tex = new GlTex2D(MULTISAMPX * w, MULTISAMPY * h,
                             0, GL_LINEAR, GL_CLAMP, GL_RGBA);
    allocate(w, h);
}

// -----------------------------------------------------------------------------

Render_context_cu::~Render_context_cu()
{
    _pbo_color->cuda_unregister();
    _pbo_depth->cuda_unregister();
    delete _pbo_color;
    delete _pbo_depth;
    delete _frame_tex;
    delete _peeler;

    Cuda_utils::free_d( _d_img_buffer      );
    Cuda_utils::free_d( _d_bloom_buffer    );
    Cuda_utils::free_d( _d_rendu_buf       );
    Cuda_utils::free_d( _d_rendu_depth_buf );
}

// -----------------------------------------------------------------------------

void Render_context_cu::reshape(int w, int h)
{
    allocate(w, h);
}

// -----------------------------------------------------------------------------

void Render_context_cu::allocate(int width, int height)
{
    _width  = width;
    _height = height;

    _frame_tex->bind();
    _frame_tex->set_size(MULTISAMPX * width, MULTISAMPY * height);
    _frame_tex->allocate(GL_UNSIGNED_BYTE, GL_RGBA);
    GlTex2D::unbind();

    if(_pbo_color != 0) _pbo_color->cuda_unregister();
    if(_pbo_depth != 0) _pbo_depth->cuda_unregister();

    /*
    delete _pbo_color;
    delete _pbo_depth;

    _pbo_color = new BufferObject<GL_PIXEL_UNPACK_BUFFER>(MULTISAMPX*width*MULTISAMPY*height);
    _pbo_depth = new BufferObject<GL_PIXEL_UNPACK_BUFFER>(width*height);
    */

    if( _pbo_color != 0 ) _pbo_color->set_data(MULTISAMPX*width*MULTISAMPY*height, 0);
    else                  _pbo_color = new GlBuffer_obj<GLint>(MULTISAMPX*width*MULTISAMPY*height, GL_PIXEL_UNPACK_BUFFER);

    if(_pbo_depth != 0) _pbo_depth->set_data(width*height, 0);
    else                _pbo_depth = new GlBuffer_obj<GLuint>(width*height, GL_PIXEL_UNPACK_BUFFER);

    // Register pbos
    _pbo_color->cuda_register();
    _pbo_depth->cuda_register();

    Cuda_utils::free_d( _d_img_buffer      );
    Cuda_utils::free_d( _d_bloom_buffer    );
    Cuda_utils::free_d( _d_rendu_buf       );
    Cuda_utils::free_d( _d_rendu_depth_buf );

    Cuda_utils::malloc_d(_d_img_buffer     , width * MULTISAMPX * height * MULTISAMPY * 2 );
    Cuda_utils::malloc_d(_d_bloom_buffer   , width * MULTISAMPX * height * MULTISAMPY * 2 );
    Cuda_utils::malloc_d(_d_rendu_buf      , width * MULTISAMPX * height * MULTISAMPY     );
    Cuda_utils::malloc_d(_d_rendu_depth_buf, width * height);

    _peeler->reinit_depth_peeling(width, height);
}

// END Render_context_cu =======================================================

// -----------------------------------------------------------------------------

#include "toolbox/timer.hpp"

/// Raytrace the implicit scene. and draw it with openGL onto a quad
/// @return false if the raytracing is complete.
bool raytrace(Render_context_cu* ctx, const Camera* cam)
{
    using namespace Cuda_ctrl;
    Color cl = _color.get(Color_ctrl::BACKGROUND);
    float4 cl_color = {cl.r, cl.g, cl.b, cl.a};
    const int width  = ctx->width();
    const int height = ctx->height();

    bool refresh = false;
    int* d_img_buf = 0;
    unsigned* d_depth   = 0;
    ctx->pbo_color()->cuda_map_to( d_img_buf );
    ctx->pbo_depth()->cuda_map_to( d_depth   );
    if( ctx->_raytrace && ctx->_skeleton)
    {
        bool prog = _display._progressive_raytracing;

        if( ctx->_rest_pose ) g_skel->reset();
        refresh = !Raytracing::raytrace_implicit(*cam,
                                                 ctx->d_render_buff(), ctx->d_depth_buff(),
                                                 d_img_buf  , d_depth,
                                                 width, height, prog);
        if( ctx->_rest_pose ) g_skel->unreset();

        ctx->pbo_color()->cuda_unmap();
        ctx->pbo_depth()->cuda_unmap();

        ctx->pbo_color()->bind();
        ctx->frame_tex()->bind();
        ctx->frame_tex()->allocate(GL_UNSIGNED_BYTE, GL_RGBA);
        ctx->pbo_color()->unbind();

        EnableProgram();
        draw_quad();
        DisableProgram();
    }
    else
    {
        clean_pbos(d_img_buf, d_depth, width, height, cl_color);
        ctx->pbo_color()->cuda_unmap();
        ctx->pbo_depth()->cuda_unmap();
    }
    return refresh;
}

// -----------------------------------------------------------------------------

void draw_one_BBox(const Bbox3 &bbox){
    Point3 pmin = bbox.pmin;
    Point3 pmax = bbox.pmax;
    glBegin(GL_LINE_STRIP);
        glVertex3f( pmin.x, pmin.y, pmin.z );
        glVertex3f( pmin.x, pmin.y, pmax.z );
        glVertex3f( pmin.x, pmax.y, pmax.z );
        glVertex3f( pmin.x, pmax.y, pmin.z );
        glVertex3f( pmin.x, pmin.y, pmin.z );
        glVertex3f( pmax.x, pmin.y, pmin.z );
        glVertex3f( pmax.x, pmin.y, pmax.z );
        glVertex3f( pmax.x, pmax.y, pmax.z );
        glVertex3f( pmax.x, pmax.y, pmin.z );
        glVertex3f( pmax.x, pmin.y, pmin.z );
    glAssert( glEnd() );
    glBegin(GL_LINE_STRIP);
        glVertex3f( pmin.x, pmin.y, pmax.z );
        glVertex3f( pmax.x, pmin.y, pmax.z );
        glVertex3f( pmax.x, pmax.y, pmax.z );
        glVertex3f( pmin.x, pmax.y, pmax.z );
        glVertex3f( pmin.x, pmax.y, pmin.z );
        glVertex3f( pmax.x, pmax.y, pmin.z );
    glAssert( glEnd() );

}

// -----------------------------------------------------------------------------

bool display_loop(Render_context_cu* ctx, const Camera* cam)
{
    using namespace Cuda_ctrl;
    const int width  = ctx->width();
    const int height = ctx->height();
    assert(cam->width()  == width  );
    assert(cam->height() == height );

    Color cl = _color.get(Color_ctrl::BACKGROUND);
    float4 cl_color = {cl.r, cl.g, cl.b, cl.a};

    glAssert( glClearColor(cl_color.x, cl_color.y, cl_color.z, cl_color.w) );
    glAssert( glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT) );

    ////////////////////////////////
    // Raytrace implicit surfaces //
    ////////////////////////////////
    bool refresh = false;

#if 0
    Color c = Color::pseudo_rand(/*Cuda_ctrl::_debug._nb_step*/ 14);
    // Original Blue
    g_ray_material.A  = Vec3(c.r, c.g, c.b) * 0.3f;
    g_ray_material.Kd = Vec3(1.f, 1.f, 1.f);
    g_ray_material.Ks = Vec3(0.2f, 0.2f, 0.2f);
    g_ray_material.sh = 20.f;
    //g_ray_material.Kd = Vec3(255.f/255.f, 218.f/255.f, 185.f/255.f);
    //g_ray_material.Kd = Vec3(0.9f, 0.83f, 0.9f);
    //g_ray_material.Kd = Vec3(c.r, c.g, c.b);
#else
    g_ray_material.A  = Vec3::unit_scale() * 0.1f;
    g_ray_material.Kd = Vec3(0.34f, 0.6f, 1.f);
    g_ray_material.Ks = Vec3(0.4f , 0.4f, 0.4f);
    g_ray_material.sh = 15.f;
#endif

    refresh = raytrace(ctx, cam);

    /////////////////////////////
    // Setup projection matrix //
    /////////////////////////////

    glAssert( glMatrixMode(GL_PROJECTION) );
    glAssert( glLoadIdentity() );
    glViewport(0, 0, width, height);
    cam->gl_mult_projection();
    glAssert( glMatrixMode(GL_MODELVIEW) );
    glLoadIdentity();

    float _light0_ambient [4] = { 0.2f, 0.2f, 0.2f, 1.0f };
    float _light0_diffuse [4] = { 1.0f, 1.0f, 1.0f, 1.0f };
    float _light0_specular[4] = { 1.0f, 1.0f, 1.0f, 1.0f };
    float _light0_position[4] = { 0.0f, 0.0f, 0.0f, 1.0f };

//    glEnable(GL_LIGHTING);
//    glEnable(GL_LIGHT0);
    glLightfv(GL_LIGHT0, GL_AMBIENT , _light0_ambient );
    glLightfv(GL_LIGHT0, GL_DIFFUSE , _light0_diffuse );
    glLightfv(GL_LIGHT0, GL_SPECULAR, _light0_specular);
    glLightfv(GL_LIGHT0, GL_POSITION, _light0_position);
    GL_CHECK_ERRORS();

    cam->lookat();


    ///////////////////////////////
    // Draw mesh with the raster //
    ///////////////////////////////

#if 1
    if(ctx->_plain_phong)
    {
        glAssert( glEnable(GL_DEPTH_TEST) );
        // Draw depth of the implicit surface (copy is slow as hell though)
        if( ctx->_raytrace )
        {
            ctx->pbo_depth()->bind();
            glAssert( glColorMask(GL_FALSE, GL_FALSE, GL_FALSE, GL_FALSE) );
            glAssert( glDrawPixels(width, height,GL_DEPTH_COMPONENT,GL_FLOAT,0) );
            glAssert( glColorMask(GL_TRUE, GL_TRUE, GL_TRUE, GL_TRUE) );
            ctx->pbo_depth()->unbind();
        }

        glColor3f(1.f, 1.f, 1.f);
        EMesh::Material mat;
        mat.setup_opengl_materials();

        if(ctx->_rest_pose)
            draw_mesh(*g_mesh, *g_animesh->get_vbo_rest_pose(), *g_animesh->get_nbo_rest_pose(), ctx->_textures);
        else
            draw_mesh(*g_mesh, g_mesh->_mesh_gl._vbo, g_mesh->_mesh_gl._normals_bo, ctx->_textures);

        glAssert( glDisable(GL_DEPTH_TEST) );

        if(_display._ssao) redraw_with_ssao();
    }
    else
    {
        RenderFuncWireframe rfunc(cam, ctx);

        ctx->peeler()->set_render_func(&rfunc);
        ctx->peeler()->set_background(width, height, ctx->pbo_color(), ctx->pbo_depth());
        ctx->peeler()->peel( _display._transparency );

        glAssert( glEnable(GL_DEPTH_TEST) );
        glAssert( glClear(GL_DEPTH_BUFFER_BIT) );

        // Draw depth of the depth peeling
        ctx->pbo_depth()->bind();
        glAssert( glColorMask(GL_FALSE, GL_FALSE, GL_FALSE, GL_FALSE) );
        glAssert( glDrawPixels(width, height,GL_DEPTH_COMPONENT,GL_FLOAT,0) );
        ctx->pbo_depth()->unbind();


        glAssert( glPolygonOffset(1.1f, 4.f) );
        glAssert( glEnable(GL_POLYGON_OFFSET_FILL) );
        rfunc.f();
        glAssert( glDisable(GL_POLYGON_OFFSET_FILL) );



        glAssert( glColorMask(GL_TRUE, GL_TRUE, GL_TRUE, GL_TRUE) );
        RenderFuncWireframe::render( cam, ctx );
        glAssert( glDisable(GL_DEPTH_TEST) );
    }
#endif

    ////////////////////////////////////
    // various drawing mostly sprites //
    ////////////////////////////////////
#if 1
    if(Cuda_ctrl::_debug._draw_grid_skeleton)
        Skeleton_env::draw_grids();

    if( !ctx->_plain_phong )
    {
        GLEnabledSave save_tex  (GL_TEXTURE_2D, true, false);
        GLEnabledSave save_light(GL_LIGHTING  , true, false);

        int s = Cuda_ctrl::_operators.get_display_size();
        if( _operators.get_display_controller() ) draw_controller();
        if( _operators.get_display_operator()   ) draw_operator(321, 0, s, s);
    }


    if(g_save_anim && g_animesh != 0 && g_anim_cache != 0)
    {
        std::vector<float> mesh_frame;
        g_animesh->get_anim_vertices_aifo(mesh_frame);

        if(mesh_frame.size() > 0)
            g_anim_cache->add_frame(&(mesh_frame[0]));
    }
#endif

    return refresh;
}
